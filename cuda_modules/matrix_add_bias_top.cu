
#include <hip/hip_runtime.h>
/***************************************************
 * Module that adds a new row at the top of the matrix with all ones
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_add_bias_top.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_add_bias_top.cu
 *
 **************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixAddBiasTop(double* C, double* A, int width, int resW, int resH, int resultSize)
{
	int x = threadIdx.x + (blockIdx.x * resW);
	int y = threadIdx.y + (blockIdx.y * resH);
	int resultPos = y * width + x;

	if (resultPos < resultSize && x < width) {
		if (y == 0) {
			C[resultPos] = 1;
		} else {
			C[resultPos] = A[resultPos - width];
		}
	}
}

#ifdef __cplusplus
}
#endif
