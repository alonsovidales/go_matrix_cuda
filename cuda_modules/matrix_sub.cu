
#include <hip/hip_runtime.h>
/***************************************************
 * Module for matrix substraction
 * Author: Alonso Vidales <alonso.vidales@tras2.es>
 *
 * To be compiled with nvcc -ptx matrix_sub.cu
 * Debug: nvcc -arch=sm_20 -ptx matrix_sub.cu
 *
 **************************************************/

//#include <stdio.h>

#ifdef __cplusplus
extern "C" {
#endif

// CUDA Kernel
__global__ void matrixSub(double* C, double* A, double* B, int width, int resW, int resH, int resultSize)
{
	int x = threadIdx.x + (blockIdx.x * resW);
	int y = threadIdx.y + (blockIdx.y * resH);
	int resultPos = y * width + x;

	if (resultPos < resultSize && x < width) {
		C[resultPos] = A[resultPos] - B[resultPos];
		//printf("Block %d - %d, thread %d - %d Val: %f\n", x, y, threadIdx.x, threadIdx.y, C[resultPos]);
	}
}

#ifdef __cplusplus
}
#endif
